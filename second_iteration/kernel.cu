#include "hip/hip_runtime.h"
//
// Created by servant-of-scietia on 9/27/24.
//

#include "kernel.cuh"
#include <iostream>

__global__ void myKernel(int* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 10) {
        data[idx] *= 2;
    }
}

void launchKernel(int* data, int size) {
    int* d_data;
    hipMalloc((void**)&d_data, size * sizeof(int));
    hipMemcpy(d_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    myKernel<<<1, size>>>(d_data);

    hipMemcpy(data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_data);
}

